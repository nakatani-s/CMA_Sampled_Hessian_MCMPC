#include "hip/hip_runtime.h"
/*
  Least Squares Method: LSM 最小二乗法
  QuadraticHyperPlane:  QHP ２次超平面
*/ 

#include<iostream>
#include <stdio.h>
#include <fstream>
#include <math.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>
#include <errno.h>
#include <string.h>
#include <unistd.h>
#include <iomanip>

#include "include/params.cuh"
#include "include/init.cuh"
#include "include/DataStructure.cuh"
#include "include/MCMPC.cuh"
#include "include/LSM_QuadHyperPlane.cuh"
#include "include/Matrix.cuh"
#include "include/costFunction.cuh"
#include "include/CovarianceMatrixAdaptation.cuh"

#define CHECK(call)                                                  \
{                                                                    \
    const hipError_t error = call;                                  \
    if (error != hipSuccess)                                        \
    {                                                                \
        printf("Error: %s:%d, ", __FILE__, __LINE__);                \
        printf("code:%d, reason: %s\n", error,                       \
                hipGetErrorString(error));                          \
        exit(1);                                                     \
    }                                                                \
}
#define CHECK_CUBLAS(call,str)                                                        \
{                                                                                     \
    if ( call != HIPBLAS_STATUS_SUCCESS)                                               \
    {                                                                                 \
        printf("CUBLAS Error: %s : %s %d\n", str, __FILE__, __LINE__);                \
        exit(1);                                                                      \
    }                                                                                 \
}

#define CHECK_CUSOLVER(call,str)                                                      \
{                                                                                     \
    if ( call != HIPSOLVER_STATUS_SUCCESS)                                             \
    {                                                                                 \
        printf("CUBLAS Error: %s : %s %d\n", str, __FILE__, __LINE__);                \
        exit(1);                                                                      \
    }                                                                                 \
}





int main(int argc, char **argv)
{
    /* 行列演算ライブラリを使用するために宣言 */
    hipsolverHandle_t cusolverH = NULL;
    hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR; 
    CHECK_CUSOLVER( hipsolverDnCreate(&cusolverH),"Failed to Create cusolver handle");

    hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;

    /* データ書き込み用ファイルの定義 */
    FILE *fp;
    time_t timeValue;
    struct tm *timeObject;
    time( &timeValue );
    timeObject = localtime( &timeValue );
    char filename1[35];
    sprintf(filename1,"data_system_%d%d_%d%d.txt",timeObject->tm_mon + 1, timeObject->tm_mday, timeObject->tm_hour,timeObject->tm_min);
    fp = fopen(filename1,"w");


    /* ホスト・デバイスで使用するベクトルの宣言 */
    float hostParams[DIM_OF_PARAMETERS], hostState[DIM_OF_STATES], hostConstraint[NUM_OF_CONSTRAINTS], hostWeightMatrix[DIM_OF_WEIGHT_MATRIX];
    float *deviceParams, *deviceState, *deviceConstraint, *deviceWeightMatrix;
    initialize_host_vector(hostParams, hostState, hostConstraint, hostWeightMatrix);
    hipMalloc(&deviceParams, sizeof(float) * DIM_OF_PARAMETERS);
    hipMalloc(&deviceState, sizeof(float) * DIM_OF_STATES);
    hipMalloc(&deviceConstraint, sizeof(float) * NUM_OF_CONSTRAINTS);
    hipMalloc(&deviceWeightMatrix, sizeof(float) * DIM_OF_WEIGHT_MATRIX);
    hipMemcpy(deviceParams, hostParams, sizeof(float) * DIM_OF_PARAMETERS, hipMemcpyHostToDevice);
    hipMemcpy(deviceState, hostState, sizeof(float) * DIM_OF_STATES, hipMemcpyHostToDevice);
    hipMemcpy(deviceConstraint, hostConstraint, sizeof(float) * NUM_OF_CONSTRAINTS, hipMemcpyHostToDevice);
    hipMemcpy(deviceWeightMatrix, hostWeightMatrix, sizeof(float)* DIM_OF_WEIGHT_MATRIX, hipMemcpyHostToDevice);

    /* GPUの設定用パラメータ */
    unsigned int numBlocks, randomBlocks, randomNums, Blocks, dimHessian, numUnknownParamQHP, numUnknownParamHessian;
    unsigned int paramsSizeQuadHyperPlane;
    randomNums = NUM_OF_SAMPLES * (DIM_OF_INPUT + 1) * HORIZON;
    randomBlocks = countBlocks(randomNums, THREAD_PER_BLOCKS);
    numBlocks = countBlocks(NUM_OF_SAMPLES, THREAD_PER_BLOCKS);
    Blocks = numBlocks;
    dimHessian = HORIZON * HORIZON;
    // numUnknownParamQHP = count_QHP_Parameters( HORIZON );
    numUnknownParamQHP = sizeOfParaboloidElements;
    numUnknownParamHessian = numUnknownParamQHP - (HORIZON + 1);
    paramsSizeQuadHyperPlane = numUnknownParamQHP; //ホライズンの大きさに併せて、局所サンプルのサイズを決定
    paramsSizeQuadHyperPlane = paramsSizeQuadHyperPlane + addTermForLSM;
    dim3 block(2,2);
    dim3 grid((numUnknownParamQHP + block.x - 1)/ block.x, (numUnknownParamQHP + block.y -1) / block.y);
    printf("#NumBlocks = %d\n", numBlocks);
    printf("#NumBlocks = %d\n", numUnknownParamQHP);

    /* GPUで乱数生成するために使用する乱数の種 */
    hiprandState *deviceRandomSeed;
    hipMalloc((void **)&deviceRandomSeed, randomNums * sizeof(hiprandState));
    setup_kernel<<<NUM_OF_SAMPLES, (DIM_OF_INPUT + 1) * HORIZON>>>(deviceRandomSeed, rand());
    hipDeviceSynchronize();

    /* sort用入力格納構造体の宣言と初期化 */
    InputVector *deviceInputSeq, *hostInputSeq, *deviceEliteInputSeq;
    hostInputSeq = (InputVector*)malloc(sizeof(InputVector) * NUM_OF_ELITES);
    hipMalloc(&deviceEliteInputSeq, sizeof(InputVector) * NUM_OF_ELITES);
    hipMalloc(&deviceInputSeq, sizeof(InputVector) * NUM_OF_SAMPLES);
    // init_Input_vector<<<NUM_OF_SAMPLES, 1>>>(deviceInputSeq, 0.0f);

    /* 2次超平面のパラメータ行列/ベクトル　（←最適値計算用に準備） */
    float *Hessian, *HessElements, *transGmatrix, *Hvector, *invGmHessSsymm /*, *Grad*/;
    hipMalloc(&Hessian, sizeof(float) * dimHessian );
    hipMalloc(&transGmatrix, sizeof(float) * dimHessian); /* Ans = -2 * G^T * Hessian * Hvector の　G^T  */
    hipMalloc(&Hvector, sizeof(float) * HORIZON ); /* Ans = -2 * G^T * Hessian * Hvector の　Hvector */
    hipMalloc(&invGmHessSsymm, sizeof(float) * dimHessian);
    // hipMalloc(&HessElements, sizeof(float) * numUnknownParamHessian);
    hipMalloc(&HessElements, sizeof(float) * numUnknownParamQHP );
    // hipMalloc(&Grad, sizeof(float) * HORIZON);
    /* 最小二乗法で2次超平面を求める際に使用 */
    float *Gmatrix, *invGmatrix, *Rvector, *ansRvector;
    CHECK(hipMalloc(&Rvector, sizeof(float) * numUnknownParamQHP));
    CHECK(hipMalloc(&ansRvector, sizeof(float) * numUnknownParamQHP));
    CHECK(hipMalloc(&Gmatrix, sizeof(float) * numUnknownParamQHP * numUnknownParamQHP));
    CHECK(hipMalloc(&invGmatrix, sizeof(float) * numUnknownParamQHP * numUnknownParamQHP) ); //elementsSize_QuadHyperPlaneMatrix = paramsSize_QuadHyperPlane * paramsSize_QuadHyperPlane
    //assert(hipSuccess == cudaStat2);
    QuadHyperPlane *deviceQuadHyPl;
    hipMalloc(&deviceQuadHyPl, sizeof(QuadHyperPlane) * paramsSizeQuadHyperPlane); //当面はブロック数分リサンプル　( HORIZON < Blocks < GPUコア数 で設計)
    unsigned int qhpBlocks;
    // qhpBlocks = countBlocks(numUnknownParamQHP, THREAD_PER_BLOCKS);
    qhpBlocks = countBlocks(paramsSizeQuadHyperPlane, THREAD_PER_BLOCKS);
    printf("#qhpblocks = %d\n", qhpBlocks);
    // float *KVALUE_MATRIX, *HESSIAN_MATRIX;
    // KVALUE_MATRIX = (float *)malloc(sizeof(float)*numUnknownParamQHP * numUnknownParamQHP);
    // HESSIAN_MATRIX = (float *)malloc(sizeof(float)*dimHessian);
    //KVALUE_MATRIX = (float *)malloc(sizeof(float)*dimHessian);
    // 行列演算ライブラリ使用用に定義
    const int m_RMatrix = numUnknownParamQHP;
    printf("#NumBlocks = %d\n", m_RMatrix);
    // const int lda_RMatrix = m_RMatrix;
    int work_size, work_size_season2;
    float *work_space, *work_space_season2;
    int *devInfo;
    int *devInfo_season2;
    hipMalloc ((void**)&devInfo_season2, sizeof(int));
    hipblasHandle_t handle_cublas = 0;
    hipblasCreate(&handle_cublas);
    float alpha;
    float beta;
    alpha = 1.0f;
    beta = 0.0f;
    hipMalloc ((void**)&devInfo, sizeof(int));

    /* Variables for CMA-ES */
    float /* *hostCov,*/ *deviceCov, *deviceSquareCov, *deviceEigDiag;
    float *deviceCovEig;
    float *d_work, *d_ws_Hess;
    int lwork = 0;
    // hostCov = (float*)malloc(sizeof(float) * HORIZON * HORIZON);
    CHECK(hipMalloc(&deviceCovEig, sizeof(float) * HORIZON));
    CHECK(hipMalloc(&deviceCov, sizeof(float) * HORIZON * HORIZON));
    CHECK(hipMalloc(&deviceSquareCov, sizeof(float) * HORIZON * HORIZON));
    CHECK(hipMalloc(&deviceEigDiag, sizeof(float) * HORIZON * HORIZON));


    /* thrust使用のためのホスト/デバイスベクトル */
    thrust::host_vector<int> indices_host_vec( NUM_OF_SAMPLES );
    thrust::device_vector<int> indices_device_vec = indices_host_vec;
    thrust::host_vector<float> sort_key_host_vec( NUM_OF_SAMPLES );
    thrust::device_vector<float> sort_key_device_vec = sort_key_host_vec; 
    
    /* 推定入力のプロット・データ転送用 */
    float *hostData, *deviceData;
    hostData = (float *)malloc(sizeof(float) * HORIZON);
    hipMalloc(&deviceData, sizeof(float) * HORIZON);
    for(int i = 0; i < HORIZON; i++){
        hostData[i] = 0.0f;
    }
    hipMemcpy(deviceData, hostData, sizeof(float) * HORIZON, hipMemcpyHostToDevice);
    
    // float variance;
    /* 制御ループ */
    float est_input = 0.0f;
    float MCMPC_U, Proposed_U;
    float costFromMCMPC, costFromQHPMethod;
    costFromMCMPC = 0.0f;
    costFromQHPMethod = 0.0f;
    float vars;

    int counter;
    float process_gpu_time, procedure_all_time;
    clock_t start_t, stop_t;
    hipEvent_t start, stop;

    for(int t = 0; t < TIME; t++){
        shift_Input_vec( hostData );
        hipMemcpy(deviceData, hostData, sizeof(float) * HORIZON, hipMemcpyHostToDevice);
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start, 0);
        start_t = clock();
        for(int repeat = 0; repeat < ITERATIONS; repeat++){
            if(repeat < ITERATIONS - 1){
                /* サンプルベースニュートンメソッドの初期値を決定するMCMPC */
                vars = powf(0.95,repeat) * variance; 
                MCMPC_Crat_and_SinglePole<<<numBlocks, THREAD_PER_BLOCKS>>>(deviceState, deviceRandomSeed, deviceData, deviceInputSeq, vars, deviceParams, deviceConstraint, deviceWeightMatrix,
                    thrust::raw_pointer_cast( sort_key_device_vec.data() ));
                /*MCMPC_Simple_NonLinear_Example<<<numBlocks, THREAD_PER_BLOCKS>>>(deviceState, deviceRandomSeed, deviceData, deviceInputSeq, variance, deviceParams, deviceConstraint, deviceWeightMatrix,
                    thrust::raw_pointer_cast( sort_key_device_vec.data() ));*/
                hipDeviceSynchronize();
                thrust::sequence(indices_device_vec.begin(), indices_device_vec.end());
                thrust::sort_by_key(sort_key_device_vec.begin(), sort_key_device_vec.end(), indices_device_vec.begin());
            
                /* エリートサンプル分の入力・コスト値をコールバックする関数 */ 
                callback_elite_sample<<<NUM_OF_ELITES, 1>>>(deviceEliteInputSeq, deviceInputSeq, thrust::raw_pointer_cast(indices_device_vec.data()));
                hipDeviceSynchronize();
                hipMemcpy(hostInputSeq, deviceEliteInputSeq, sizeof(InputVector) * NUM_OF_ELITES, hipMemcpyDeviceToHost);
                weighted_mean(hostInputSeq, NUM_OF_ELITES, hostData);
                MCMPC_U = hostData[0];
            
                CHECK(hipMemcpy(deviceData, hostData, sizeof(float) * HORIZON, hipMemcpyHostToDevice));
            
                costFromMCMPC = calc_Cost_Cart_and_SinglePole(hostData, hostState, hostParams, hostConstraint, hostWeightMatrix);
                // printf("%dth MCMPC estimation ended\n", t*repeat);
            }else{
                vars = powf(0.95,repeat) * variance; 
                // 分散共分散行列の更新(CMA-ES)
                // cpy_Previous_CovarianceMatrix<<<HORIZON, HORIZON>>>(deviceSquareCov, deviceCov);
                setup_Identity_Matrix<<<HORIZON, HORIZON>>>(deviceSquareCov);  // deviceSquareCov を　単位行列化
                make_Covariance_Matrix<<< HORIZON, HORIZON>>>(deviceCov, deviceSquareCov, deviceData, deviceInputSeq, thrust::raw_pointer_cast(indices_device_vec.data()), 3 * NUM_OF_ELITES);
                result_CMA_rank_mu_Update<<<HORIZON, HORIZON>>>(deviceCov, deviceSquareCov, c_learning_rate);
                // 分散共分散行列の固有値をベクトルに返すcuSOLVER関数
                CHECK_CUSOLVER(hipsolverDnSsyevd_bufferSize(cusolverH, jobz, uplo, HORIZON, deviceCov, HORIZON, deviceCovEig, &lwork),"Failed to compute size of d_workspace for clc");
                CHECK( hipMalloc((void**)&d_work, sizeof(float) * lwork) );
                // 固有値(deviceCovEig)と固有ベクトル(deviceCov)を取得するコマンド
                CHECK_CUSOLVER( hipsolverDnSsyevd(cusolverH, jobz, uplo, HORIZON, deviceCov, HORIZON, deviceCovEig, d_work, lwork, devInfo), "Failed to get eigenValues of Covariance Matrix" );
                CHECK( hipDeviceSynchronize() );
                // 固有値(deviceCovEig)を（昇順に）対角に並べた対角行列（deviceEigDiag）を生成する関数の実行
                // make_Eigen_Diagonal_Matrix<<<HORIZON, HORIZON>>>(deviceEigDiag, deviceCovEig);
                make_SqrtEigen_Diagonal_Matrix<<<HORIZON, HORIZON>>>(deviceEigDiag, deviceCovEig);
                
                // 
                // W(deviceEigDiag) = P^t(deviceCov) V(deviceEigDiag) を計算する関数  
                pwr_matrix_answerLater<<<HORIZON, HORIZON>>>(deviceCov, deviceEigDiag);
                // 正規直交固有ベクトル(deviceCov)を転置した行列(deviceSquareCov)を作成する関数の実行
                LSM_QHP_transpose<<<HORIZON, HORIZON>>>(deviceSquareCov, deviceCov);
                pwr_matrix_answerLater<<<HORIZON, HORIZON>>>(deviceEigDiag, deviceSquareCov);
                // vars ← 共分散のスケーリングは、当初固定、事後、最終反復時のMCコスト／　最初の反復時のMCコストを採用予定
                // CMAを用いた並列シミュレーション用の関数の作成　←　ここから作成する　（2021.5.12）
                /*CMAMCMPC_Cart_and_SinglePole<<<numBlocks, THREAD_PER_BLOCKS>>>(deviceState, deviceRandomSeed, deviceSquareCov, deviceData, deviceInputSeq, neighborVar, deviceParams, deviceConstraint, 
                    deviceWeightMatrix, thrust::raw_pointer_cast( sort_key_device_vec.data() ));*/
                CMAMCMPC_Cart_and_SinglePole<<<numBlocks, THREAD_PER_BLOCKS>>>(deviceState, deviceRandomSeed, deviceSquareCov, deviceData, deviceInputSeq, 1.0f, deviceParams, deviceConstraint, 
                    deviceWeightMatrix, thrust::raw_pointer_cast( sort_key_device_vec.data() ));
                hipDeviceSynchronize();
                thrust::sequence(indices_device_vec.begin(), indices_device_vec.end());
                thrust::sort_by_key(sort_key_device_vec.begin(), sort_key_device_vec.end(), indices_device_vec.begin());
                
                /* エリートサンプル分の入力・コスト値をコールバックする関数 */ 
                callback_elite_sample<<<NUM_OF_ELITES, 1>>>(deviceEliteInputSeq, deviceInputSeq, thrust::raw_pointer_cast(indices_device_vec.data()));
                hipDeviceSynchronize();
                hipMemcpy(hostInputSeq, deviceEliteInputSeq, sizeof(InputVector) * NUM_OF_ELITES, hipMemcpyDeviceToHost);
                weighted_mean(hostInputSeq, NUM_OF_ELITES, hostData);
                MCMPC_U = hostData[0];
                
                CHECK(hipMemcpy(deviceData, hostData, sizeof(float) * HORIZON, hipMemcpyHostToDevice));
                
                costFromMCMPC = calc_Cost_Cart_and_SinglePole(hostData, hostState, hostParams, hostConstraint, hostWeightMatrix);
                // printf("%dth MCMPC estimation ended\n", t*repeat);

                // ↓↓↓↓　以降の関数は、近傍探索から
                /* 推定値近傍をサンプル・評価する関数 */
                /*CMAMCMPC_Cart_and_SinglePole<<<numBlocks, THREAD_PER_BLOCKS>>>(deviceState, deviceRandomSeed, deviceSquareCov, deviceData, deviceInputSeq, 1.0f, deviceParams, deviceConstraint, 
                    deviceWeightMatrix, thrust::raw_pointer_cast( sort_key_device_vec.data() ));*/
                MCMPC_Crat_and_SinglePole<<<numBlocks, THREAD_PER_BLOCKS>>>(deviceState, deviceRandomSeed, deviceData, deviceInputSeq, neighborVar, deviceParams, deviceConstraint, deviceWeightMatrix,
                    thrust::raw_pointer_cast( sort_key_device_vec.data() ));
                /*MCMPC_Simple_NonLinear_Example<<<numBlocks, THREAD_PER_BLOCKS>>>(deviceState, deviceRandomSeed, deviceData, deviceInputSeq, neighborVar, deviceParams, deviceConstraint, deviceWeightMatrix,
                    thrust::raw_pointer_cast( sort_key_device_vec.data() ));*/
                hipDeviceSynchronize();
                thrust::sequence(indices_device_vec.begin(), indices_device_vec.end());
                thrust::sort_by_key(sort_key_device_vec.begin(), sort_key_device_vec.end(), indices_device_vec.begin());
                /*device_QuadHyPlに，最小二乗法の左辺(column)と右辺の行列(テンソル積)計算用のベクトル(tensor)を格納する*/
                // printf("hoge here l 185\n");
                LSM_QHP_make_tensor_vector<<<qhpBlocks, THREAD_PER_BLOCKS>>>(deviceQuadHyPl, deviceInputSeq, thrust::raw_pointer_cast( indices_device_vec.data() ));
                // printf("hoge here l 187\n");
                hipDeviceSynchronize();
            /* Gmatrix に正規行列（最小二乗法で使用する逆行列の逆行列）*/ 
                if(numUnknownParamQHP > 1024){
                    LSM_QHP_make_regular_matrix_over_ThreadPerBlockLimit<<<grid,block>>>(Gmatrix, deviceQuadHyPl, paramsSizeQuadHyperPlane, numUnknownParamQHP);
                }else{
                    LSM_QHP_make_regular_matrix<<<numUnknownParamQHP,numUnknownParamQHP>>>(Gmatrix, deviceQuadHyPl, paramsSizeQuadHyperPlane);
                }
                hipDeviceSynchronize();
                // printf("hoge here l 193\n");

                // 最小二乗法の結果（ヘシアンの要素＋勾配＋定数）
                LSM_QHP_make_regular_vector<<<numUnknownParamQHP,1>>>(Rvector, deviceQuadHyPl, paramsSizeQuadHyperPlane);
                hipDeviceSynchronize();

            /* Gmatrixの逆行列を計算 */
                CHECK_CUSOLVER( hipsolverDnSpotrf_bufferSize(cusolverH, uplo, m_RMatrix, Gmatrix, m_RMatrix, &work_size), "Failed to get bufferSize");
                CHECK( hipMalloc((void**)&work_space, sizeof(float)*work_size));
	            //hipGetErrorString(cudaStat1);
                //assert(hipSuccess == cudaStat1);
                CHECK_CUSOLVER(hipsolverDnSpotrf(cusolverH, uplo, m_RMatrix, Gmatrix, m_RMatrix , work_space, work_size, devInfo), "Failed to inverse operation for G");
                
                // 逆行列を取得するための単位行列の生成
                if(numUnknownParamQHP > 1024){
                    setup_Identity_Matrix_overMaxThread<<<grid, block>>>(invGmatrix, numUnknownParamQHP); 
                }else{
                    setup_Identity_Matrix<<<numUnknownParamQHP, numUnknownParamQHP>>>(invGmatrix); // invGmatrixを単位行列に変換
                }
                hipDeviceSynchronize();
                CHECK_CUSOLVER( hipsolverDnSpotrs(cusolverH, uplo, m_RMatrix, m_RMatrix , Gmatrix, m_RMatrix, invGmatrix, m_RMatrix, devInfo), "Failed to get inverse Matrix G");

                //LSM_QHP_get_reslt_all_elements<<<numUnknownParamQHP,1>>>(HessElements, invGmatrix, Rvector);
                /* 最小二乗法の行列演算　ansRvector = invGmatrix * Rvector を計算 */ 
                CHECK_CUBLAS( hipblasSgemv(handle_cublas, HIPBLAS_OP_N, m_RMatrix, m_RMatrix, &alpha, invGmatrix, m_RMatrix, Rvector, 1, &beta, ansRvector , 1), "Failed to get Estimate Input Sequences");

                //assert(  cublas_status == HIPBLAS_STATUS_SUCCESS );
                LSM_QHP_get_reslt_all_elements<<<numUnknownParamHessian,1>>>(HessElements, ansRvector); //numUnknownParamHessian これが大きすぎる?
                hipDeviceSynchronize();
                LSM_QHP_get_Hessian_Result<<<HORIZON, HORIZON>>>( Hessian, HessElements);
                CHECK( hipDeviceSynchronize() );
                // 行列の転置を計算、ここでは、上三角行列から下三角行列を生成している
                // 行列が特殊型（上三角or下三角など）ない場合は、下の関数で行列の転置を計算できる．
                LSM_QHP_transpose<<<HORIZON, HORIZON>>>(transGmatrix, Hessian);
                hipDeviceSynchronize();
                // 上三角行列と下三角行列の要素を調べ、対称行列となるように結合
                LSM_QHP_make_symmetric<<<HORIZON, HORIZON>>>(transGmatrix, Hessian);
                // hipMemcpy(hostCov, transGmatrix, sizeof(float) * dimHessian, hipMemcpyDeviceToHost);
                // printMatrix(HORIZON,HORIZON,hostCov, HORIZON, "HESSIAN");
                // ヘッシアンの計算まで終了
                //LSM_Hessian_To_Positive_Symmetric<<<HORIZON, HORIZON>>>(transGmatrix);

                /* -2*Hessian * b^T の b^Tベクトルを作成 (Hvector　←　b^T) */
                LSM_QHP_make_bVector<<<HORIZON, 1>>>(Hvector, ansRvector, numUnknownParamHessian);

                multiply_matrix<<<HORIZON, HORIZON>>>(Hessian, 2.0f, transGmatrix);
                // 逆行列の計算方法を変更
#ifdef INVERSE_OPERATION_USING_EIGENVALUE
                CHECK_CUSOLVER(hipsolverDnSsyevd_bufferSize(cusolverH, jobz, uplo, HORIZON, Hessian, HORIZON, deviceCovEig, &lwork),"Failed to compute size of d_workspace for get eigen value for Hessian");
                CHECK( hipMalloc((void**)&d_ws_Hess, sizeof(float) * lwork) );
                CHECK_CUSOLVER( hipsolverDnSsyevd(cusolverH, jobz, uplo, HORIZON, Hessian, HORIZON, deviceCovEig, d_ws_Hess, lwork, devInfo), "Failed to compute Eigen values of Hessian");
                CHECK( hipDeviceSynchronize() );
                make_InverseEigen_Diagonal_Matrix<<<HORIZON,  HORIZON>>>(transGmatrix, deviceCovEig);
                pwr_matrix_answerLater<<<HORIZON, HORIZON>>>(Hessian, transGmatrix);
                CHECK( hipDeviceSynchronize() );
                LSM_QHP_transpose<<<HORIZON, HORIZON>>>(invGmHessSsymm, Hessian);
                CHECK( hipDeviceSynchronize() );
                pwr_matrix_answerLater<<<HORIZON, HORIZON>>>(transGmatrix, invGmHessSsymm);
                CHECK( hipDeviceSynchronize() );
#else
                CHECK_CUSOLVER(hipsolverDnSpotrf_bufferSize(cusolverH, uplo, HORIZON, Hessian, HORIZON, &work_size_season2),"Failed to get bufferSize of Hessian");
                CHECK( hipMalloc((void**)&work_space_season2, sizeof(float)*work_size_season2) );
  
                CHECK_CUSOLVER(hipsolverDnSpotrf(cusolverH, uplo, HORIZON, Hessian, HORIZON, work_space_season2, work_size_season2, devInfo_season2), "Failed to inverse operation");
            
                setup_Identity_Matrix<<<HORIZON, HORIZON>>>(invGmHessSsymm);
                hipDeviceSynchronize();
                CHECK_CUSOLVER(hipsolverDnSpotrs(cusolverH, uplo, HORIZON, HORIZON, Hessian, HORIZON, invGmHessSsymm, HORIZON, devInfo_season2), "Failed to get inverse Matrix of H");
                // hipMemcpy(HESSIAN_MATRIX, invGmHessSsymm, sizeof(float) * dimHessian, hipMemcpyDeviceToHost);
#endif
                multiply_matrix<<<HORIZON, HORIZON>>>(transGmatrix, -1.0f, invGmHessSsymm);

                copy_inputSequences<<<numBlocks, THREAD_PER_BLOCKS>>>(deviceInputSeq, deviceData);
                CHECK_CUBLAS(hipblasSgemv(handle_cublas, HIPBLAS_OP_N, HORIZON, HORIZON, &alpha, transGmatrix, HORIZON, Hvector, 1, &beta,  deviceData, 1),"Failed to get Result");
                //cublas_status = hipblasSgemv(handle_cublas, HIPBLAS_OP_N, HORIZON, HORIZON, &alpha, invGmHessSsymm, HORIZON, Hvector, 1, &beta,  deviceData, 1);
                hipMemcpy(hostData, deviceData, sizeof(float) * HORIZON, hipMemcpyDeviceToHost);
                //costFromQHPMethod = calc_Cost_Simple_NonLinear_Example(hostData, hostState,  hostParams, hostWeightMatrix);
                costFromQHPMethod = calc_Cost_Cart_and_SinglePole(hostData, hostState, hostParams, hostConstraint, hostWeightMatrix);
                Proposed_U = hostData[0];
            }

        }
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&process_gpu_time, start, stop);
        stop_t = clock();
        procedure_all_time = stop_t - start_t;
        // 推定入力列の先頭をコピー
        if(costFromMCMPC < costFromQHPMethod || isnan(costFromQHPMethod)){
	        est_input = MCMPC_U;
            counter = 0;
	    }else{
            est_input = Proposed_U;
            counter = 1;
        }
        Runge_kutta_45_for_Secondary_system(hostState, est_input, hostParams, interval);
        /*float hostDiffState[DIM_OF_STATES] = { };
        calc_nonLinear_example(hostState, est_input, hostParams, hostDiffState);
        for(int k = 0; k < DIM_OF_STATES; k++){
            hostState[k] = hostState[k] + (interval * hostDiffState[k]);
        }*/
        hipMemcpy(deviceState, hostState, sizeof(float) * DIM_OF_STATES, hipMemcpyHostToDevice);
        fprintf(fp,"%f %f %f %f %f %f %f %f %f %f %f %f %f %d\n", interval * t, est_input, MCMPC_U, Proposed_U, hostState[0], hostState[1], hostState[2], hostState[3], costFromMCMPC, costFromQHPMethod, costFromMCMPC - costFromQHPMethod, process_gpu_time/1000,procedure_all_time / CLOCKS_PER_SEC, counter);
        printf("u == %f MCMPC == %f  Proposed == %f  MCMPC - Proposed == %f\n", est_input,  costFromMCMPC, costFromQHPMethod, costFromMCMPC - costFromQHPMethod);
    }

    if(cusolverH) hipsolverDnDestroy(cusolverH);
    if(handle_cublas) hipblasDestroy(handle_cublas);
    fclose(fp);
    hipDeviceReset();
    printf("%s\n", hipGetErrorString(hipGetLastError()));
    return 0;
}
